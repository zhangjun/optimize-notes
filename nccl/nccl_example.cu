#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


void all_reduce()
{
  printf("\n#####AllReduce Begin#### \n");

  ncclComm_t comms[2];


  //managing 4 devices
  int nDev = 2;
  int size = 1024;
  int devs[2] = { 0, 1};


  //allocating and initializing device buffers
  int** sendbuff = (int**)malloc(nDev * sizeof(int*));
  int** recvbuff = (int**)malloc(nDev * sizeof(int*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);


  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(int)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(int)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(int)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(int)));
    CUDACHECK(hipStreamCreate(s+i));
  }


  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));


   //calling NCCL communication API. Group API is required when using
   //multiple devices per thread
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i)
    NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclInt, ncclSum,
        comms[i], s[i]));
  NCCLCHECK(ncclGroupEnd());



  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }


 {
     int idx = 0;
     int* result = (int*)malloc(size * sizeof(int));
     CUDACHECK(hipSetDevice(idx));
     CUDACHECK(hipMemcpy((void*)result, (void*)recvbuff[idx], size * sizeof(int), hipMemcpyDeviceToHost));
     int* p = (int*)result;
     printf("check result: %d,%d\n", p[0], p[1]);
     free(result);
 }

  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }


  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);


  printf("AllReduce Success \n");
}

void broadcast() {
  printf("\n####Broadcast Begin#### \n");

  ncclComm_t comms[2];

  int nDev = 2;
  int devs[2] = {0, 1};
  int size = 32 * 1024 * 1024;


  int** buf = (int**)malloc(sizeof(int*) * nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t) * nDev);

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipMalloc(buf + i, size * sizeof(int));
      hipMemset(buf[i], i, size * sizeof(int));
      hipStreamCreate(s + i);
  }

  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  ncclGroupStart();
  for(int i = 0; i < nDev; i ++) {
      // sendbuf only used in root rank
      ncclBroadcast((const void*)buf[i], (void*)buf[i], size, ncclFloat, 1, comms[i], s[i]);
  }
  ncclGroupEnd();

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipStreamSynchronize(s[i]);
  }

  {
     int idx = 0;
     int* result = (int*)malloc(size * sizeof(int));
     CUDACHECK(hipSetDevice(idx));
     CUDACHECK(hipMemcpy((void*)result, (void*)buf[idx], size * sizeof(int), hipMemcpyDeviceToHost));
     int* p = (int*)result;
     printf("check result: %d,%d\n", p[0], p[1]);
     free(result);
  }

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipFree(buf[i]);
  }

  for(int i = 0; i < nDev; i ++) {
      ncclCommDestroy(comms[i]);
  }

  printf("Broadcast Success \n");
}

void reduce() {
  printf("\n####Reduce Begin#### \n");

  ncclComm_t comms[2];

  int nDev = 2;
  int devs[2] = {0, 1};
  int size = 32 * 1024 * 1024;


  int** buf = (int**)malloc(sizeof(int*) * nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t) * nDev);

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipMalloc(buf + i, size * sizeof(int));
      hipMemset(buf[i], 1, size * sizeof(int));
      hipStreamCreate(s + i);
  }

  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  ncclGroupStart();
  for(int i = 0; i < nDev; i ++) {
      // recvbuf only used in root rank
      ncclReduce((const void*)buf[i], (void*)buf[i], size, ncclInt, ncclSum, 0, comms[i], s[i]);
  }
  ncclGroupEnd();

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipStreamSynchronize(s[i]);
  }

  {
     int idx = 0;
     int* result = (int*)malloc(size * sizeof(int));
     CUDACHECK(hipSetDevice(idx));
     CUDACHECK(hipMemcpy((void*)result, (void*)buf[idx], size * sizeof(int), hipMemcpyDeviceToHost));
     int* p = (int*)result;
     printf("check result: %d,%d\n", p[0], p[1]);
     free(result);
  }

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipFree(buf[i]);
  }

  for(int i = 0; i < nDev; i ++) {
      ncclCommDestroy(comms[i]);
  }

  printf("Broadcast Success \n");
}


// all_gather = reduce_scatter + all_reduce
void all_gather() {
  printf("\n####Allgather Begin#### \n");

  ncclComm_t comms[2];

  int nDev = 2;
  int devs[2] = {0, 1};
  int size = 32 * 1024 * 1024;


  int** buf = (int**)malloc(sizeof(int*) * nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t) * nDev);

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipMalloc(buf + i, nDev * size * sizeof(int));
      hipMemset(buf[i], i, nDev * size * sizeof(int));
      hipStreamCreate(s + i);
  }

  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  ncclGroupStart();
  for(int i = 0; i < nDev; i ++) {
      // recvbuf only used in root rank
      ncclAllGather((const void*)buf[i], (void*)buf[i], size, ncclInt, comms[i], s[i]);
  }
  ncclGroupEnd();

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipStreamSynchronize(s[i]);
  }

  {
     int idx = 0;
     int* result = (int*)malloc(size * sizeof(int) * nDev);
     CUDACHECK(hipSetDevice(idx));
     CUDACHECK(hipMemcpy((void*)result, (void*)buf[idx], nDev * size * sizeof(int), hipMemcpyDeviceToHost));
     int* p = (int*)result;
     printf("check result: %d,%d, %d,%d\n", p[0], p[1], p[size], p[size + 1]);
     free(result);
  }

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipFree(buf[i]);
  }

  for(int i = 0; i < nDev; i ++) {
      ncclCommDestroy(comms[i]);
  }

  printf("Allgather Success \n");
}


void reduce_scatter() {
  printf("\n####ReduceScatter Begin#### \n");

  ncclComm_t comms[2];

  int nDev = 2;
  int devs[2] = {0, 1};
  int size = 32 * 1024 * 1024;


  int** buf = (int**)malloc(sizeof(int*) * nDev);
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t) * nDev);

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipMalloc(buf + i, nDev * size * sizeof(int));
      hipMemset(buf[i], i, nDev * size * sizeof(int));
      hipStreamCreate(s + i);
  }

  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  ncclGroupStart();
  for(int i = 0; i < nDev; i ++) {
      // recvbuf only used in root rank
      ncclReduceScatter((const void*)buf[i], (void*)buf[i], size, ncclInt, ncclSum, comms[i], s[i]);
  }
  ncclGroupEnd();

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipStreamSynchronize(s[i]);
  }

  {
     int idx = 0;
     int* result = (int*)malloc(size * nDev * sizeof(int));
     CUDACHECK(hipSetDevice(idx));
     CUDACHECK(hipMemcpy((void*)result, (void*)buf[idx], nDev * size * sizeof(int), hipMemcpyDeviceToHost));
     int* p = (int*)result;
     printf("check result: %d,%d %d,%d\n", p[0], p[1], p[size], p[size + 1]);
     free(result);
  }

  for(int i = 0; i < nDev; i ++) {
      hipSetDevice(i);
      hipFree(buf[i]);
  }

  for(int i = 0; i < nDev; i ++) {
      ncclCommDestroy(comms[i]);
  }

  printf("ReduceScatter Success \n");
}



int main(int argc, char* argv[]) {

  all_reduce();  

  broadcast();

  reduce();

  all_gather();

  reduce_scatter();

  return 0;
}
