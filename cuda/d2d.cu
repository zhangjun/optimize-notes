#include <stdio.h>

#include "hip/hip_runtime.h"
#include "cuda_util.h"

int main() {
    const int count = 1024 * 1024;
    int *first = NULL, *second = NULL;

    int device_count;
    hipGetDeviceCount(&device_count);
    printf("device count: %d\n", device_count);


    {
        CUDACHECK(hipSetDevice(0));
        CUDACHECK(hipMalloc((void**)&first, sizeof(int) * count));
        CUDACHECK(hipMemset(first, 0, sizeof(int) * count)); 
    }

    {
        CUDACHECK(hipSetDevice(1));
        CUDACHECK(hipMalloc((void**)&second, sizeof(int) * count));
        CUDACHECK(hipMemset(second, 1, sizeof(int) * count)); 
    }

    hipMemcpy(first, second, sizeof(int) * count, hipMemcpyDeviceToDevice);

    {
        CUDACHECK(hipSetDevice(1));
        int *data = (int*)malloc(sizeof(int) * count);
        memset(data, 0, sizeof(int) * count); 
        CUDACHECK(hipMemcpy(data, first, sizeof(int) * count, hipMemcpyDeviceToHost)); 
        printf("data: %d\n", data[0]);

        free(data);
    }

    hipFree(first);
    hipFree(second);

    return 0;

}
