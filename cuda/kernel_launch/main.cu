#include "hip/hip_runtime.h"
#include <iostream>

__global__ void empty() {}

int main() {
  int device_id = 0;
  hipSetDevice(device_id);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  int repeat = 1000;

  hipEventRecord(start, stream);
  for (int i = 0; i < repeat; ++i) {
    empty<<<1, 1>>>();
  }
  hipEventRecord(end, stream);
  hipEventSynchronize(end);
  float elapse_time = 0.0f;
  hipEventElapsedTime(&elapse_time, start, end);
  std::cout << "launch latency: " << elapse_time / (1.0f * repeat) << " ms."
            << std::endl;
  return 0;
}